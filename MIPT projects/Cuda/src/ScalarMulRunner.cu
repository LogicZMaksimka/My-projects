#include <ScalarMulRunner.cuh>

#include <CommonKernels.cuh>

#include <KernelMul.cuh>

#include <ScalarMul.cuh>

#include <cstdio>

float ScalarMulTwoReductions(int num_elements, float *vector1, float *vector2, int block_size) {
  int blocks_count = (num_elements + block_size - 1) / block_size;

  float *device_vector1;
  float *device_vector2;
  float *device_block_sum;

  float *host_block_sum = (float *) calloc(blocks_count, sizeof(float));

  hipMalloc(&device_vector1, num_elements * sizeof(float));
  hipMalloc(&device_vector2, num_elements * sizeof(float));
  hipMalloc(&device_block_sum, blocks_count * sizeof(float));

  hipMemcpy(device_vector1, vector1, num_elements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_vector2, vector2, num_elements * sizeof(float), hipMemcpyHostToDevice);

  ScalarMulBlock<<<blocks_count, block_size, block_size * sizeof(float)>>>(num_elements,
                                                                           device_vector1,
                                                                           device_vector2,
                                                                           device_block_sum);

  hipDeviceSynchronize();
  hipMemcpy(host_block_sum, device_block_sum, blocks_count * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_vector1);
  hipFree(device_vector2);
  hipFree(device_block_sum);

  float total_sum = 0.0f;
  for (int i = 0; i < blocks_count; ++i) {
    total_sum += host_block_sum[i];
  }
  return total_sum;
}

float ScalarMulSumPlusReduction(int num_elements, float *vector1, float *vector2, int block_size) {
  int blocks_count = (num_elements + block_size - 1) / block_size;

  float *device_vector1;
  float *device_vector2;
  float *device_product_vector;
  float *device_block_sum;

  float *host_block_sum = (float *) calloc(blocks_count, sizeof(float));

  hipMalloc(&device_vector1, num_elements * sizeof(float));
  hipMalloc(&device_vector2, num_elements * sizeof(float));
  hipMalloc(&device_product_vector, num_elements * sizeof(float));
  hipMalloc(&device_block_sum, blocks_count * sizeof(float));

  hipMemcpy(device_vector1, vector1, num_elements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_vector2, vector2, num_elements * sizeof(float), hipMemcpyHostToDevice);

  KernelMul<<<blocks_count, block_size>>>(num_elements, device_vector1, device_vector2, device_product_vector);
  ArrayBlockSum<<<blocks_count, block_size, block_size * sizeof(float)>>>(num_elements,
                                                                          device_product_vector,
                                                                          device_block_sum);

  hipMemcpy(host_block_sum, device_block_sum, blocks_count * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_vector1);
  hipFree(device_vector2);
  hipFree(device_product_vector);
  hipFree(device_block_sum);

  float total_sum = 0.0f;
  for (int i = 0; i < blocks_count; ++i) {
    total_sum += host_block_sum[i];
  }
  return total_sum;
}
