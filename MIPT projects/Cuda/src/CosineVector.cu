#include "hip/hip_runtime.h"
#include <CosineVector.cuh>

#include <ScalarMulRunner.cuh>

float CosineVector(int num_elements, float* vector1, float* vector2, int block_size) {
  float dot_product = ScalarMulTwoReductions(num_elements, vector1, vector2, block_size);
  float vector1_length = sqrt(ScalarMulTwoReductions(num_elements, vector1, vector1, block_size));
  float vector2_length = sqrt(ScalarMulTwoReductions(num_elements, vector2, vector2, block_size));

  return dot_product / (vector1_length * vector2_length);
}
