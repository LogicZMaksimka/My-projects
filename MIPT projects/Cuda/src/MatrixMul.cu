#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

__device__ void SetMatrixElement(float *matrix, int width, int row, int column, float element) {
  matrix[row * width + column] = element;
}

__device__ float GetMatrixElement(float *matrix, int width, int row, int column) {
  return matrix[row * width + column];
}

// NECESSARY REQUIREMENTS:
// 1) blockDim.x * gridDim.x >= max(height_A, height_B)
// 2) blockDim.y * gridDim.y >= max(width_A, width_B)
// 3) blockDim.x == blockDim.

// Algorithm:
// 1) Split matrix A and B into sub matrices with size (blockDim.x, blockDim.y)
// 2) For every block load row and column sub matrices in shared memory
// 3) For every block calculate sum over A_sub_matrix * B_sub_matrix
__global__ void MatrixMul(int height_A,
                          int width_A,
                          int width_B,
                          float *matrix_A,
                          float *matrix_B,
                          float *matrix_result) {
  int block_row = blockIdx.x;
  int block_column = blockIdx.y;
  int block_size = blockDim.x; // blockDim.x == blockDim.y

  // can allocate only 1 extern __shared__ array
  // so we need to split it into 2 arrays
  extern __shared__ float shared_data[];
  float *A_sub_matrix = shared_data;
  float *B_sub_matrix = (float *) &shared_data[block_size * block_size];

  float result_element = 0.0f;
  for (int block_num = 0; block_num < gridDim.y; ++block_num) {
    // load sub matrices in shared memory
    int A_element_row = block_row * block_size + threadIdx.x;
    int A_element_column = block_num * block_size + threadIdx.y;
    int B_element_row = block_num * block_size + threadIdx.x;
    int B_element_column = block_column * block_size + threadIdx.y;

    SetMatrixElement(A_sub_matrix, block_size, threadIdx.x, threadIdx.y, 0.0f);
    SetMatrixElement(B_sub_matrix, block_size, threadIdx.x, threadIdx.y, 0.0f);
    if (A_element_row < height_A && A_element_column < width_A) {
      float A_element = GetMatrixElement(matrix_A, width_A, A_element_row, A_element_column);
      SetMatrixElement(A_sub_matrix, block_size, threadIdx.x, threadIdx.y, A_element);
    }
    if (B_element_row < width_A && B_element_column < width_B) {
      float B_element = GetMatrixElement(matrix_B, width_B, B_element_row, B_element_column);
      SetMatrixElement(B_sub_matrix, block_size, threadIdx.x, threadIdx.y, B_element);
    }
    __syncthreads();

    // calculate result_element
    for (int i = 0; i < block_size; ++i) {
      float A_sub_matrix_element = GetMatrixElement(A_sub_matrix, block_size, threadIdx.x, i);
      float B_sub_matrix_element = GetMatrixElement(B_sub_matrix, block_size, i, threadIdx.y);
      result_element += A_sub_matrix_element * B_sub_matrix_element;
    }
    __syncthreads();
  }

  int result_row = blockIdx.x * blockDim.x + threadIdx.x;
  int result_column = blockIdx.y * blockDim.y + threadIdx.y;
  if (result_row < height_A && result_column < width_B) {
    SetMatrixElement(matrix_result, width_B, result_row, result_column, result_element);
  }
}