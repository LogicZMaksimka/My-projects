#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

// NECESSARY REQUIREMENT: blockDim.x * gridDim.x >= height
__global__ void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
  int row_index = blockIdx.x * blockDim.x + threadIdx.x;
  result[row_index] = 0.0f;
  if(row_index < height) {
    for(int column_index = 0; column_index < width; ++column_index) {
     result[row_index] += matrix[row_index * width + column_index] * vector[column_index];
    }
  }
}

