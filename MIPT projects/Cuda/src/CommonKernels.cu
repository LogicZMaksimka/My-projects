#include "hip/hip_runtime.h"
#include <CommonKernels.cuh>

// NECESSARY REQUIREMENT: blockDim.x * gridDim.x >= array_size
__global__ void ArrayBlockSum(int array_size, float *array, float *block_sum) {
  extern __shared__ float shared_array[];
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(index < array_size) {
    shared_array[threadIdx.x] = array[index];
  } else {
    shared_array[threadIdx.x] = 0.0f;
  }
  __syncthreads();

  for(int half_size = blockDim.x / 2; half_size > 0; half_size /= 2) {
    if(threadIdx.x < half_size) {
      shared_array[threadIdx.x] += shared_array[threadIdx.x + half_size];
    }
    __syncthreads();
  }

  if(threadIdx.x == 0) {
     block_sum[blockIdx.x] = shared_array[0];
  }
}
