#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

// NECESSARY REQUIREMENTS: blockDim.x * gridDim.x >= height; blockDim.y * gridDim.y >= width
__global__ void KernelMatrixAdd(int height, int width, size_t pitch, float* A, float* B, float* result) {
  int row_index = blockIdx.x * blockDim.x + threadIdx.x;
  int column_index = blockIdx.y * blockDim.y + threadIdx.y;
  if(row_index < height && column_index < width) {
    float *A_row = (float *)((char *)A + row_index * pitch);
    float *B_row = (float *)((char *)B + row_index * pitch);
    float *result_row = (float *)((char *)result + row_index * pitch);
    result_row[column_index] = A_row[column_index] + B_row[column_index];
  }
}
