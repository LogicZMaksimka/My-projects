#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

/*
 * Calculates scalar multiplication for block
 */
__global__
void ScalarMulBlock(int numElements, float* vector1, float* vector2, float *result) {
  extern __shared__ float thread_sum[];
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float sum = 0.0f;
  for(int i = index; i < numElements; i += stride) {
    sum += vector1[i] * vector2[i];
  }
  thread_sum[threadIdx.x] = sum;
  __syncthreads();

  //reduction
  for(int half_size = blockDim.x / 2; half_size > 0; half_size /= 2) {
    if(threadIdx.x < half_size) {
      thread_sum[threadIdx.x] += thread_sum[threadIdx.x + half_size];
    }
    __syncthreads();
  }
  if(threadIdx.x == 0) {
    result[blockIdx.x] = thread_sum[0];
  }
}

