#include <KernelMatrixAdd.cuh>
#include <cstdio>

void AddMatrices(int height, int width, float *host_matrix_x, float *host_matrix_y, float *host_matrix_sum, int block_size) {
  float *device_matrix_x;
  float *device_matrix_y;
  float *device_matrix_sum;

  size_t pitch;

  size_t host_pitch = width * sizeof(float);

  hipMallocPitch(&device_matrix_x, &pitch, width * sizeof(float), height);
  hipMallocPitch(&device_matrix_y, &pitch, width * sizeof(float), height);
  hipMallocPitch(&device_matrix_sum, &pitch, width * sizeof(float), height);

  hipMemcpy2D(device_matrix_x,
               pitch,
               host_matrix_x,
               host_pitch,
               width * sizeof(float),
               height,
               hipMemcpyHostToDevice);
  hipMemcpy2D(device_matrix_y,
               pitch,
               host_matrix_y,
               host_pitch,
               width * sizeof(float),
               height,
               hipMemcpyHostToDevice);

  dim3 block_dim(block_size, block_size);
  dim3 grid_dim((height + block_dim.x - 1) / block_dim.x, (width + block_dim.y - 1) / block_dim.y);

  KernelMatrixAdd<<<grid_dim, block_dim>>>(height,
                                           width,
                                           pitch,
                                           device_matrix_x,
                                           device_matrix_y,
                                           device_matrix_sum);
  hipMemcpy2D(host_matrix_sum,
               host_pitch,
               device_matrix_sum,
               pitch,
               width * sizeof(float),
               height,
               hipMemcpyDeviceToHost);

  hipFree(device_matrix_x);
  hipFree(device_matrix_y);
  hipFree(device_matrix_sum);
}

int main() {
  FILE *file = fopen("../time_measurements/add_matrices", "w");
  for (int matrix_size = 1 << 5; matrix_size < 1 << 14; matrix_size *= 2) {
    for (int block_size = 2; block_size <= 1 << 5; block_size *= 2) {
      printf("matrix_size = %d block_size = %d\n", matrix_size, block_size);
      int height = matrix_size;
      int width = matrix_size;
      int matrix_size = width * height;

      float *host_matrix_x = (float *) calloc(matrix_size, sizeof(float));
      float *host_matrix_y = (float *) calloc(matrix_size, sizeof(float));
      float *host_matrix_sum = (float *) calloc(matrix_size, sizeof(float));

      for (int i = 0; i < matrix_size; ++i) {
        host_matrix_x[i] = (float) rand() / RAND_MAX;
        host_matrix_y[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      AddMatrices(height, width, host_matrix_x, host_matrix_y, host_matrix_sum, block_size);

      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds = 0;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      float time_in_seconds = time_in_milliseconds / 1000.0;
      fprintf(file, "%.6f,", time_in_seconds);

      free(host_matrix_x);
      free(host_matrix_y);
      free(host_matrix_sum);
    }
    fprintf(file, "\n");
  }

  fclose(file);
  return 0;
}
