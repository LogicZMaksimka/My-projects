#include "KernelAdd.cuh"
#include <stdio.h>
#include <stdlib.h>

void AddVectors(int array_size, float *host_array_x, float *host_array_y, float *host_array_sum, int block_size) {
  int array_memory_size = array_size * sizeof(float);

  float *device_array_x;
  float *device_array_y;
  float *device_array_sum;

  hipMalloc(&device_array_x, array_memory_size);
  hipMalloc(&device_array_y, array_memory_size);
  hipMalloc(&device_array_sum, array_memory_size);

  hipMemcpy(device_array_x, host_array_x, array_memory_size, hipMemcpyHostToDevice);
  hipMemcpy(device_array_y, host_array_y, array_memory_size, hipMemcpyHostToDevice);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int blocks_count = (array_size + block_size - 1) / block_size;

  KernelAdd<<<blocks_count, block_size>>>(array_size, device_array_x, device_array_y, device_array_sum);
  hipMemcpy(host_array_sum, device_array_sum, array_memory_size, hipMemcpyDeviceToHost);

  hipFree(device_array_x);
  hipFree(device_array_y);
  hipFree(device_array_sum);
}

int main() {
  FILE *file = fopen("../time_measurements/vectors_sum", "w");
  for (int array_size = 1 << 11; array_size < 1 << 28; array_size *= 8) {
    for (int block_size = 4; block_size <= 1 << 10; block_size *= 4) {
      printf("array_size = %d block_size = %d\n", array_size, block_size);
      float *host_array_x = (float *) calloc(array_size, sizeof(float));
      float *host_array_y = (float *) calloc(array_size, sizeof(float));
      float *host_array_sum = (float *) calloc(array_size, sizeof(float));

      for (int i = 0; i < array_size; ++i) {
        host_array_x[i] = (float) rand() / RAND_MAX;
        host_array_y[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      AddVectors(array_size, host_array_x, host_array_y, host_array_sum, block_size);

      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds = 0;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      float time_in_seconds = time_in_milliseconds / 1000.0;
      fprintf(file, "%.6f,", time_in_seconds);

      free(host_array_x);
      free(host_array_y);
      free(host_array_sum);
    }
    fprintf(file, "\n");
  }

  fclose(file);
  return 0;
}
