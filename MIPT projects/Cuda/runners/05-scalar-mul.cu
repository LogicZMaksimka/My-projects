#include <ScalarMulRunner.cuh>
#include <cstdio>

int main() {
  FILE *file_2reductions = fopen("../time_measurements/dot_product_2_reductions", "w");
  FILE *file_mul_plus_reduction = fopen("../time_measurements/dot_product_mul_plus_reduction", "w");
  for (int vector_length = 1 << 11; vector_length < 1 << 28; vector_length *= 8) {
    for (int block_size = 4; block_size <= 1 << 10; block_size *= 4) {
      printf("vector_length = %d block_size = %d\n", vector_length, block_size);
      float *host_vector1 = (float *) calloc(vector_length, sizeof(float));
      float *host_vector2 = (float *) calloc(vector_length, sizeof(float));

      for (int i = 0; i < vector_length; ++i) {
        host_vector1[i] = (float) rand() / RAND_MAX;
        host_vector2[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      //___________________________________________________________________________
      hipEventRecord(start);
      float dot_product1 = ScalarMulSumPlusReduction(vector_length, host_vector1, host_vector2, block_size);
      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      fprintf(file_mul_plus_reduction, "%.6f,", time_in_milliseconds / 1000.0);
      //___________________________________________________________________________

      hipEventRecord(start);
      float dot_product2 = ScalarMulTwoReductions(vector_length, host_vector1, host_vector2, block_size);
      hipEventRecord(stop);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      fprintf(file_2reductions, "%.6f,", time_in_milliseconds / 1000.0);
      //___________________________________________________________________________

      free(host_vector1);
      free(host_vector2);
    }
    fprintf(file_2reductions, "\n");
    fprintf(file_mul_plus_reduction, "\n");
  }
  fclose(file_2reductions);
  fclose(file_mul_plus_reduction);
}

