#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

#include <cmath>
#include <cstdio>

void MultiplyMatrices(int height_A,
                      int width_A,
                      int width_B,
                      float *host_matrix_A,
                      float *host_matrix_B,
                      float *host_matrix_result,
                      int block_size) {
  int height_B = width_A;
  int A_elements_count = height_A * width_A;
  int B_elements_count = height_B * width_B;
  int result_elements_count = height_A * width_B;

  float *device_matrix_A;
  float *device_matrix_B;
  float *device_matrix_result;

  hipMalloc(&device_matrix_A, A_elements_count * sizeof(float));
  hipMalloc(&device_matrix_B, B_elements_count * sizeof(float));
  hipMalloc(&device_matrix_result, result_elements_count * sizeof(float));

  hipMemcpy(device_matrix_A, host_matrix_A, A_elements_count * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_matrix_B, host_matrix_B, B_elements_count * sizeof(float), hipMemcpyHostToDevice);

  dim3 block_dim(block_size, block_size);
  dim3 grid_dim((max(height_A, height_B) + block_dim.x - 1) / block_dim.x,
                (max(width_A, width_B) + block_dim.y - 1) / block_dim.y);
  size_t shared_memory_size = (2 * block_dim.x * block_dim.y + 1) * sizeof(float);

  MatrixMul<<<grid_dim, block_dim, shared_memory_size>>>(
      height_A,
      width_A,
      width_B,
      device_matrix_A,
      device_matrix_B,
      device_matrix_result);
  hipMemcpy(host_matrix_result, device_matrix_result, result_elements_count * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_matrix_A);
  hipFree(device_matrix_B);
  hipFree(device_matrix_result);
}

int main() {
  FILE *file = fopen("../time_measurements/product_of_matrices", "w");
  for (int matrix_size = 1 << 5; matrix_size < 1 << 14; matrix_size *= 4) {
    for (int block_size = 2; block_size <= 1 << 5; block_size *= 2) {
      printf("matrix_size = %d block_size = %d\n", matrix_size, block_size);
      int height_A = matrix_size;
      int width_A = matrix_size;
      int height_B = width_A;
      int width_B = matrix_size;

      int A_elements_count = height_A * width_A;
      int B_elements_count = height_B * width_B;
      int result_elements_count = height_A * width_B;

      float *host_matrix_A = (float *) calloc(A_elements_count, sizeof(float));
      float *host_matrix_B = (float *) calloc(B_elements_count, sizeof(float));
      float *host_matrix_result = (float *) calloc(result_elements_count, sizeof(float));

      for (int i = 0; i < A_elements_count; ++i) {
        host_matrix_A[i] = (float) rand() / RAND_MAX;
      }
      for (int i = 0; i < B_elements_count; ++i) {
        host_matrix_B[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      MultiplyMatrices(height_A, width_A, width_B, host_matrix_A, host_matrix_B, host_matrix_result, block_size);

      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      fprintf(file, "%.6f,", time_in_milliseconds / 1000.0);

      free(host_matrix_A);
      free(host_matrix_B);
      free(host_matrix_result);
    }
    fprintf(file, "\n");
  }
  fclose(file);
}

