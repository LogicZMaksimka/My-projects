#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <cstdio>

int main() {
  FILE *file = fopen("../time_measurements/cos_between_vectors", "w");
  for (int vector_length = 1 << 11; vector_length < 1 << 28; vector_length *= 8) {
    for (int block_size = 4; block_size <= 1 << 10; block_size *= 4) {
      printf("vector_length = %d block_size = %d\n", vector_length, block_size);
      float *host_vector1 = (float *) calloc(vector_length, sizeof(float));
      float *host_vector2 = (float *) calloc(vector_length, sizeof(float));

      for (int i = 0; i < vector_length; ++i) {
        host_vector1[i] = (float) rand() / RAND_MAX;
        host_vector2[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);
      float cos = CosineVector(vector_length, host_vector1, host_vector2, block_size);
      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      fprintf(file, "%.6f,", time_in_milliseconds / 1000.0);

      free(host_vector1);
      free(host_vector2);
    }
    fprintf(file, "\n");

  }
  fclose(file);
}