#include <MatrixVectorMul.cuh>
#include <cstdio>

void MultiplyMatrixByVector(int height,
                            int width,
                            float *host_matrix,
                            float *host_vector,
                            float *host_result,
                            int block_size) {
  int total_elements_count = width * height;

  float *device_matrix;
  float *device_vector;
  float *device_result;

  hipMalloc(&device_matrix, total_elements_count * sizeof(float));
  hipMalloc(&device_vector, width * sizeof(float));
  hipMalloc(&device_result, height * sizeof(float));

  hipMemcpy(device_matrix, host_matrix, total_elements_count * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_vector, host_vector, width * sizeof(float), hipMemcpyHostToDevice);

  int blocks_count = (height + block_size - 1) / block_size;

  MatrixVectorMul<<<blocks_count, block_size>>>(height, width, device_matrix, device_vector, device_result);
  hipMemcpy(host_result, device_result, height * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_matrix);
  hipFree(device_vector);
  hipFree(device_result);
}

int main() {
  FILE *file = fopen("../time_measurements/multiply_matrix_by_vector", "w");
  for (int matrix_size = 1 << 5; matrix_size < 1 << 14; matrix_size *= 2) {
    for (int block_size = 4; block_size <= 1 << 10; block_size *= 4) {
      printf("matrix_size = %d block_size = %d\n", matrix_size, block_size);
      int height = matrix_size;
      int width = matrix_size;
      int total_elements_count = width * height;

      float *host_matrix = (float *) calloc(total_elements_count, sizeof(float));
      float *host_vector = (float *) calloc(width, sizeof(float));
      float *host_result = (float *) calloc(height, sizeof(float));

      for (int i = 0; i < total_elements_count; ++i) {
        host_matrix[i] = (float) rand() / RAND_MAX;
      }
      for (int i = 0; i < width; ++i) {
        host_vector[i] = (float) rand() / RAND_MAX;
      }

      hipEvent_t start;
      hipEvent_t stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      MultiplyMatrixByVector(height, width, host_matrix, host_vector, host_result, block_size);

      hipEventRecord(stop);
      hipEventSynchronize(stop);

      float time_in_milliseconds = 0;
      hipEventElapsedTime(&time_in_milliseconds, start, stop);
      float time_in_seconds = time_in_milliseconds / 1000.0;
      fprintf(file, "%.6f,", time_in_seconds);

      free(host_matrix);
      free(host_vector);
      free(host_result);
    }
    fprintf(file, "\n");
  }

  fclose(file);
}
